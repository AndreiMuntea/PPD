#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <sal.h>
#include <malloc.h>
#include <stdio.h>


#define UNUSED_PARAMTER(x) ((void)(x))
#define MAX(a, b) (((a) > (b)) ? (a) : (b))


__device__
unsigned int
GetNumberOfDigits(
	_In_	__int64		Value
)
{
	if (Value == 0)
	{
		return 1;
	}

	int result = 0;
	for (; Value != 0; ++result, Value /= 10);

	return result;
}


__device__
__int64
Pow(
	_In_	__int64		Base,
	_In_	__int64		Power
)
{
	if (Power == 0)
	{
		return 1;
	}

	if (Power == 1)
	{
		return Base;
	}

	__int64 result = Pow(Base * Base, Power / 2);
	if (Power % 2 != 0)
	{
		result *= Base;
	}

	return result;
}


__global__
void
Karatsuba(
	_In_   __int64		FirstNumber,
	_In_   __int64		SecondNumber,
	_Out_  __int64*		Result
)
{
	// Base case
	if (FirstNumber < 10 || SecondNumber < 10)
	{
		*Result =  FirstNumber * SecondNumber;
		return;
	}

	int xLength = GetNumberOfDigits(FirstNumber);
	int yLength = GetNumberOfDigits(SecondNumber);
	int maxLength = MAX(xLength, yLength);

	int N = (maxLength / 2) + (maxLength % 2);

	__int64 multiplier = Pow(10, N);

	__int64 b = FirstNumber / multiplier;				// high part of the first number
    __int64 a = FirstNumber - (b * multiplier);			// low part of the first number

	__int64 d = SecondNumber / multiplier;				// high part of the second number
	__int64 c = SecondNumber - (d * multiplier);		// low part of the second number

	__int64* result1 = (__int64*)malloc(sizeof(__int64));
	__int64* result2 = (__int64*)malloc(sizeof(__int64));
	__int64* result3 = (__int64*)malloc(sizeof(__int64));

	Karatsuba <<<1, 1 >>> (a, c, result1);
	Karatsuba <<<1, 1 >>> (a + b, c + d, result2);
	Karatsuba <<<1, 1 >>> (b, d, result3);

	hipDeviceSynchronize();

	*Result =  *result1 + ((*result2 - *result1 - *result3) * multiplier) + (*result3 * Pow(10, 2 * N));

	free(result1);
	free(result2);
	free(result3);
}


hipError_t
Solve(
	_In_   __int64	 FirstNumber,
	_In_   __int64	 SecondNumber,
	_Out_  __int64*  Result
);

int
main(
	_In_	int		ArgumentsCount,
	_In_	char*	Arguments[]
)
{
	UNUSED_PARAMTER(ArgumentsCount);
	UNUSED_PARAMTER(Arguments);

	__int64 a = 100001;
	__int64 b = 100001;
	__int64 c = 0;

	hipError_t cudaStatus = Solve(a, b, &c);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Solve failed!");
        return 1;
    }

	fprintf(stdout, "Result of %lld * %lld = %lld\n", a, b, c);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t
Solve(
	_In_   __int64	 FirstNumber,
	_In_   __int64	 SecondNumber,
	_Out_  __int64*  Result
)
{
	__int64* dev_c = 0;
	hipError_t cudaStatus = hipSuccess;

	cudaStatus = hipMalloc((void**)&dev_c, sizeof(__int64));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

    cudaStatus = hipDeviceSetLimit(hipLimitMallocHeapSize, 1024 * 1024 * 500);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSetLimit failed!");
		goto Error;
	}

	Karatsuba<<<1, 1>>>(FirstNumber, SecondNumber, dev_c);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}


	cudaStatus = hipMemcpy(Result, dev_c, sizeof(__int64), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);

	return cudaStatus;
}